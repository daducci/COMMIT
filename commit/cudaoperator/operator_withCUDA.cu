#include "hip/hip_runtime.h"
#include "operator_withCUDA.cuh"

// ====================================================
// Textures for LUT in the GPU memory
// ====================================================
texture<float32_t, 1, hipReadModeElementType> tex_lutIC;
texture<float32_t, 1, hipReadModeElementType> tex_lutEC;
texture<float32_t, 1, hipReadModeElementType> tex_lutISO;


int checkCompatibility(int gpuID) {
    int gpuCount;
    hipError_t cudaStatus;
    
    cudaStatus = hipGetDeviceCount(&gpuCount);

    if (gpuCount <= 0 || gpuID >= gpuCount || cudaStatus != hipSuccess) return 1;

    cudaStatus = hipSetDevice(gpuID);

    if (cudaStatus != hipSuccess) return 2;

    hipDeviceProp_t gpuProperties;
    cudaStatus = hipGetDeviceProperties(&gpuProperties, gpuID);

    if (cudaStatus != hipSuccess) return 3;

    printf("\t* selected GPU...       [ %s ]\n",     gpuProperties.name);
    printf("\t* total memory...       [ %.2fGB ]\n", gpuProperties.totalGlobalMem*1e-9);
    printf("\t* compute capability... [ %d.%d ]\n",  gpuProperties.major, gpuProperties.minor);

    if(gpuProperties.major < 5) return 4;

    return 0;
}

void cudaCheckLastError()
{
    hipError_t err = hipGetLastError();

    if(err != hipSuccess){
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        exit(-1);
    }
}

void preprocessDataForGPU(uint32_t* data, int NUM_COMPARTMENTS, uint32_t* compartmentsPerBlock, uint32_t* offsetPerBlock, int NUM_BLOCKS){

    // fill arrays with zeros
    memset(compartmentsPerBlock, 0, NUM_BLOCKS * sizeof(uint32_t));
    memset(offsetPerBlock,       0, NUM_BLOCKS * sizeof(uint32_t));

    // count compartments per block
    for(int i = 0; i < NUM_COMPARTMENTS; i++)
        compartmentsPerBlock[data[i]]++;

    // calculate offset per block
    offsetPerBlock[0] = 0;
    for(int i = 1; i < NUM_BLOCKS; i++)
        offsetPerBlock[i] = offsetPerBlock[i-1] + compartmentsPerBlock[i-1];
}

int CudaLinearOperator::setDictionary(uint32_t* voxelIC, uint32_t* fiberIC, uint16_t* orienIC, float32_t* lengthIC, uint32_t* voxelEC, uint16_t* orienEC){
    
    hipError_t cudaStatus;

    uint32_t* segmentsPerBlock = (uint32_t*) malloc(nvoxels*sizeof(uint32_t));
    uint32_t* offsetPerBlock   = (uint32_t*) malloc(nvoxels*sizeof(uint32_t));

    if (segmentsPerBlock == NULL || offsetPerBlock == NULL) return -1;

    preprocessDataForGPU(voxelIC, nsegments, segmentsPerBlock, offsetPerBlock, nvoxels);

    cudaStatus = hipMalloc((void**)&gpu_segmentsPerBlockIC, nvoxels*sizeof(uint32_t));
    if (cudaStatus != hipSuccess) return 1;
    cudaStatus = hipMalloc((void**)&gpu_offsetPerBlockIC,   nvoxels*sizeof(uint32_t));
    if (cudaStatus != hipSuccess) return 1;

    cudaStatus = hipMemcpy(gpu_segmentsPerBlockIC, segmentsPerBlock, nvoxels*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) return 2;
    cudaStatus = hipMemcpy(gpu_offsetPerBlockIC,   offsetPerBlock,   nvoxels*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) return 2;

    if (npeaks > 0){
        preprocessDataForGPU(voxelEC, npeaks, segmentsPerBlock, offsetPerBlock, nvoxels);

        cudaStatus = hipMalloc((void**)&gpu_segmentsPerBlockEC, nvoxels*sizeof(uint32_t));
        if (cudaStatus != hipSuccess) return 1;
        cudaStatus = hipMalloc((void**)&gpu_offsetPerBlockEC,   nvoxels*sizeof(uint32_t));
        if (cudaStatus != hipSuccess) return 1;

        cudaStatus = hipMemcpy(gpu_segmentsPerBlockEC, segmentsPerBlock, nvoxels*sizeof(uint32_t), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) return 2;
        cudaStatus = hipMemcpy(gpu_offsetPerBlockEC,   offsetPerBlock,   nvoxels*sizeof(uint32_t), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) return 2;
    }

    free(segmentsPerBlock);
    free(offsetPerBlock);

    // alloc IC part of the dictionary in GPU
    cudaStatus = hipMalloc((void**)&gpu_voxelIC,  nsegments*sizeof(uint32_t)); 
    if (cudaStatus != hipSuccess) return 1;
    cudaStatus = hipMalloc((void**)&gpu_fiberIC,  nsegments*sizeof(uint32_t)); 
    if (cudaStatus != hipSuccess) return 1;
    cudaStatus = hipMalloc((void**)&gpu_orienIC,  nsegments*sizeof(uint16_t)); 
    if (cudaStatus != hipSuccess) return 1;
    cudaStatus = hipMalloc((void**)&gpu_lengthIC, nsegments*sizeof(float32_t));
    if (cudaStatus != hipSuccess) return 1;

    // transfer IC part of the dictionary to GPU
    cudaStatus = hipMemcpy(gpu_voxelIC,  voxelIC,  nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) return 2;
    cudaStatus = hipMemcpy(gpu_fiberIC,  fiberIC,  nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) return 2;
    cudaStatus = hipMemcpy(gpu_orienIC,  orienIC,  nsegments*sizeof(uint16_t),  hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) return 2;
    cudaStatus = hipMemcpy(gpu_lengthIC, lengthIC, nsegments*sizeof(float32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) return 2;

    if (npeaks > 0){
        // alloc EC part of the dictionary in GPU
        cudaStatus = hipMalloc((void**)&gpu_voxelEC,  npeaks*sizeof(uint32_t));
        if (cudaStatus != hipSuccess) return 1;
        cudaStatus = hipMalloc((void**)&gpu_orienEC,  npeaks*sizeof(uint16_t));
        if (cudaStatus != hipSuccess) return 1;

        // transfer EC part of the dictionary to GPU
        cudaStatus = hipMemcpy(gpu_voxelEC,  voxelEC,  npeaks*sizeof(uint32_t), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) return 2;
        cudaStatus = hipMemcpy(gpu_orienEC,  orienEC,  npeaks*sizeof(uint16_t), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) return 2;
    }

    return 0;
}

int CudaLinearOperator::setTransposeDictionary(uint32_t* TvoxelIC, uint32_t* TfiberIC, uint16_t* TorienIC, float32_t* TlengthIC){
    
    hipError_t cudaStatus;

    uint32_t*  fibersPerBlock = (uint32_t*) malloc(nfibers*sizeof(uint32_t));
    uint32_t*  offsetPerBlock = (uint32_t*) malloc(nfibers*sizeof(uint32_t));
    if(fibersPerBlock == NULL || offsetPerBlock == NULL) return -1;

    preprocessDataForGPU(TfiberIC, nsegments, fibersPerBlock, offsetPerBlock, nfibers);

    cudaStatus = hipMalloc((void**)&gpu_TfibersPerBlockIC, nfibers*sizeof(uint32_t));
    if (cudaStatus != hipSuccess) return 1;
    cudaStatus = hipMalloc((void**)&gpu_ToffsetPerBlockIC, nfibers*sizeof(uint32_t));
    if (cudaStatus != hipSuccess) return 1;

    cudaStatus = hipMemcpy(gpu_TfibersPerBlockIC, fibersPerBlock, nfibers*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) return 2;
    cudaStatus = hipMemcpy(gpu_ToffsetPerBlockIC, offsetPerBlock, nfibers*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) return 2;

    free(fibersPerBlock);
    free(offsetPerBlock);

    cudaStatus = hipMalloc((void**)&gpu_TvoxelIC,  nsegments*sizeof(uint32_t)) ;
    if (cudaStatus != hipSuccess) return 1;
    cudaStatus = hipMalloc((void**)&gpu_TfiberIC,  nsegments*sizeof(uint32_t)) ;
    if (cudaStatus != hipSuccess) return 1;
    cudaStatus = hipMalloc((void**)&gpu_TorienIC,  nsegments*sizeof(uint16_t)) ;
    if (cudaStatus != hipSuccess) return 1;
    cudaStatus = hipMalloc((void**)&gpu_TlengthIC, nsegments*sizeof(float32_t));
    if (cudaStatus != hipSuccess) return 1;

    cudaStatus = hipMemcpy(gpu_TvoxelIC,  TvoxelIC,  nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) return 2;
    cudaStatus = hipMemcpy(gpu_TfiberIC,  TfiberIC,  nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) return 2;
    cudaStatus = hipMemcpy(gpu_TorienIC,  TorienIC,  nsegments*sizeof(uint16_t),  hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) return 2;
    cudaStatus = hipMemcpy(gpu_TlengthIC, TlengthIC, nsegments*sizeof(float32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) return 2;
    
    return 0;
}

int CudaLinearOperator::setKernels(float32_t* lutIC, float32_t* lutEC, float32_t* lutISO){

    hipError_t cudaStatus;

    if (ndiameters > 0){
        cudaStatus = hipMalloc((void**)&gpu_lutIC, size_lutic*sizeof(float32_t));
        if (cudaStatus != hipSuccess) return 1;
        cudaStatus = hipMemcpy(gpu_lutIC, lutIC, size_lutic*sizeof(float32_t), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) return 2;

        tex_lutIC.addressMode[0] = hipAddressModeBorder;
        tex_lutIC.addressMode[1] = hipAddressModeBorder;
        tex_lutIC.filterMode = hipFilterModePoint;
        tex_lutIC.normalized = false;

        cudaStatus = hipBindTexture(NULL, tex_lutIC,  gpu_lutIC,  size_lutic*sizeof(float32_t));
        if (cudaStatus != hipSuccess) return 3;
    }

    if (nzeppelins > 0){
        cudaStatus = hipMalloc((void**)&gpu_lutEC,  size_lutec*sizeof(float32_t));
        if (cudaStatus != hipSuccess) return 1;
        cudaStatus = hipMemcpy(gpu_lutEC, lutEC, size_lutec*sizeof(float32_t), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) return 2;

        tex_lutEC.addressMode[0] = hipAddressModeBorder;
        tex_lutEC.addressMode[1] = hipAddressModeBorder;
        tex_lutEC.filterMode = hipFilterModePoint;
        tex_lutEC.normalized = false;

        cudaStatus = hipBindTexture(NULL, tex_lutEC,  gpu_lutEC,  size_lutec*sizeof(float32_t));
        if (cudaStatus != hipSuccess) return 3;
    }

    if (nballs > 0){
        cudaStatus = hipMalloc((void**)&gpu_lutISO, size_lutiso*sizeof(float32_t));
        if (cudaStatus != hipSuccess) return 1;
        cudaStatus = hipMemcpy(gpu_lutISO, lutISO, size_lutiso*sizeof(float32_t), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) return 2;

        tex_lutISO.addressMode[0] = hipAddressModeBorder;
        tex_lutISO.addressMode[1] = hipAddressModeBorder;
        tex_lutISO.filterMode = hipFilterModePoint;
        tex_lutISO.normalized = false;

        cudaStatus = hipBindTexture(NULL, tex_lutISO, gpu_lutISO, size_lutiso*sizeof(float32_t));
        if (cudaStatus != hipSuccess) return 3;
    }

    return 0;
}

int CudaLinearOperator::setVectors(){
    
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&gpu_x, ncols*sizeof(float64_t));
    if (cudaStatus != hipSuccess) return 1;
    cudaStatus = hipMalloc((void**)&gpu_y, nrows*sizeof(float64_t));
    if (cudaStatus != hipSuccess) return 1;
    
    return 0;
}

int CudaLinearOperator::setGlobals(){
    
    hipError_t cudaStatus;

    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(NUM_VOXELS),       &nvoxels,       sizeof(int));
    if (cudaStatus != hipSuccess) return -1;
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(NUM_FIBERS),       &nfibers,       sizeof(int));
    if (cudaStatus != hipSuccess) return -1;
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(NUM_PEAKS),        &npeaks,        sizeof(int));
    if (cudaStatus != hipSuccess) return -1;
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(NUM_ORIENTATIONS), &norientations, sizeof(int));
    if (cudaStatus != hipSuccess) return -1;
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(NUM_SAMPLES),      &nsamples,      sizeof(int));
    if (cudaStatus != hipSuccess) return -1;
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(NUM_DIAMETERS),    &ndiameters,    sizeof(int));
    if (cudaStatus != hipSuccess) return -1;
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(NUM_ZEPPELINS),    &nzeppelins,    sizeof(int));
    if (cudaStatus != hipSuccess) return -1;
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(NUM_BALLS),        &nballs,        sizeof(int));
    if (cudaStatus != hipSuccess) return -1;
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(NUM_ROWS),         &nrows,         sizeof(int));
    if (cudaStatus != hipSuccess) return -1;
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(NUM_COLS),         &ncols,         sizeof(int));
    if (cudaStatus != hipSuccess) return -1;
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(SIZE_LUTIC),       &size_lutic,    sizeof(int));
    if (cudaStatus != hipSuccess) return -1;
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(SIZE_LUTEC),       &size_lutec,    sizeof(int));
    if (cudaStatus != hipSuccess) return -1;
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(SIZE_LUTISO),      &size_lutiso,   sizeof(int));
    if (cudaStatus != hipSuccess) return -1;
    
    return 0;
}

CudaLinearOperator::CudaLinearOperator(int nsegments, int nvoxels, int nfibers, int npeaks, int norientations, int nsamples, int ndiameters, int nzeppelins, int nballs){

    this->nsegments = nsegments;
    this->nvoxels = nvoxels;
    this->nfibers = nfibers;
    this->npeaks = npeaks;
    this->norientations = norientations;
    this->nsamples = nsamples;
    this->ndiameters = ndiameters;
    this->nzeppelins = nzeppelins;   
    this->nballs = nballs;
    this->size_lutic = ndiameters*norientations*nsamples;
    this->size_lutec = nzeppelins*norientations*nsamples;
    this->size_lutiso = nballs*nsamples;
    this->nrows = nvoxels*nsamples;
    this->ncols = nfibers*ndiameters + npeaks*nzeppelins + nvoxels*nballs;
}

CudaLinearOperator::~CudaLinearOperator() {}

int CudaLinearOperator::destroy(){
    hipError_t cudaStatus;    

    cudaStatus = hipFree(gpu_voxelIC);
    cudaStatus = hipFree(gpu_fiberIC);
    cudaStatus = hipFree(gpu_orienIC);
    cudaStatus = hipFree(gpu_lengthIC);
    cudaStatus = hipFree(gpu_voxelEC);
    cudaStatus = hipFree(gpu_orienEC);
    cudaStatus = hipFree(gpu_segmentsPerBlockIC);
    cudaStatus = hipFree(gpu_offsetPerBlockIC);
    cudaStatus = hipFree(gpu_segmentsPerBlockEC);
    cudaStatus = hipFree(gpu_offsetPerBlockEC);

    cudaStatus = hipFree(gpu_TvoxelIC);
    cudaStatus = hipFree(gpu_TfiberIC);
    cudaStatus = hipFree(gpu_TorienIC);
    cudaStatus = hipFree(gpu_TlengthIC);
    cudaStatus = hipFree(gpu_TfibersPerBlockIC);
    cudaStatus = hipFree(gpu_ToffsetPerBlockIC);

    cudaStatus = hipFree(gpu_x);
    cudaStatus = hipFree(gpu_y);

    cudaStatus = hipFree(gpu_lutIC);
    cudaStatus = hipFree(gpu_lutEC);
    cudaStatus = hipFree(gpu_lutISO);
    cudaStatus = hipUnbindTexture(tex_lutIC);
    cudaStatus = hipUnbindTexture(tex_lutEC);
    cudaStatus = hipUnbindTexture(tex_lutISO);

    cudaStatus = hipDeviceReset();

    return 0;
}

void cudaCheckKernel(){
    hipError_t cudaStatus;
    
    cudaStatus = hipGetLastError();
	if(cudaStatus != hipSuccess)
        fprintf(stderr, "\t* kernel launch... [ ERROR ]: %s\n\n", hipGetErrorString(cudaStatus));
    else
        printf("\t* kernel launch... [ OK ]\n");

    cudaStatus = hipDeviceSynchronize();
	if(cudaStatus != hipSuccess)
        fprintf(stderr, "\t* hipDeviceSynchronize() after launching kernel... [ ERROR ]: %d\n", cudaStatus);
    else
        printf("\t* hipDeviceSynchronize() after launching kernel... [ OK ]\n");
}

void CudaLinearOperator::dot(float64_t* v_in, float64_t* v_out){
    
    // Copy vector x to the GPU
    hipMemcpy(gpu_x, v_in, ncols*sizeof(float64_t), hipMemcpyHostToDevice);
    //cudaCheckLastError();

    // Multiply IC part in the GPU
    multiply_Ax_ICpart<<<nvoxels, 1024>>>(gpu_voxelIC, gpu_fiberIC, gpu_orienIC, gpu_lengthIC, gpu_segmentsPerBlockIC, gpu_offsetPerBlockIC, gpu_lutIC, gpu_x, gpu_y);
    //cudaCheckLastError();

    // Multiply EC part in the GPU
    multiply_Ax_ECpart<<<nvoxels, 512>>>(gpu_voxelEC, gpu_orienEC, gpu_segmentsPerBlockEC, gpu_offsetPerBlockEC, gpu_lutEC, gpu_x, gpu_y);
    //cudaCheckLastError();

    // Multiply ISO part in the GPU
    multiply_Ax_ISOpart<<<nvoxels, 512>>>(gpu_lutISO, gpu_x, gpu_y);
    //cudaCheckLastError();

    // Copy back result to CPU
    hipMemcpy(v_out, gpu_y, nrows*sizeof(float64_t), hipMemcpyDeviceToHost);
    //cudaCheckLastError();
}

void CudaLinearOperator::Tdot(float64_t* v_in, float64_t* v_out){
    
    // Copy vector y to the GPU
    hipMemcpy(gpu_y, v_in, nrows*sizeof(float64_t), hipMemcpyHostToDevice);
    //cudaCheckLastError();

    // Multiply IC part in the GPU
    multiply_Aty_ICpart<<<nfibers, 512>>>(gpu_TvoxelIC, gpu_TfiberIC, gpu_TorienIC, gpu_TlengthIC, gpu_TfibersPerBlockIC, gpu_ToffsetPerBlockIC, gpu_lutIC, gpu_x, gpu_y);
    //cudaCheckLastError();

    // Multiply EC part in the GPU
    multiply_Aty_ECpart<<<nvoxels, 512>>>(gpu_voxelEC, gpu_orienEC, gpu_segmentsPerBlockEC, gpu_offsetPerBlockEC, gpu_lutEC, gpu_x, gpu_y);
    //cudaCheckLastError();

    // Multiply ISO part in the GPU
    multiply_Aty_ISOpart<<<nvoxels, 512>>>(gpu_lutISO, gpu_x, gpu_y);
    //cudaCheckLastError();

    // Copy back result to CPU
    hipMemcpy(v_out, gpu_x, ncols*sizeof(float64_t), hipMemcpyDeviceToHost);
    //cudaCheckLastError();
}

// ============================================================================================================================================================
// Function Kernels that are called from CPU and executed in GPU
// ============================================================================================================================================================
__global__ void multiply_Ax_ICpart(uint32_t*  voxelIDs,
                                   uint32_t*  fiberIDs,
                                   uint16_t*  orienIDs,
                                   float32_t* lengths,
                                   uint32_t*  segmentsPerBlock,
                                   uint32_t*  offsetPerBlock,
                                   float32_t* lut,
                                   float64_t* x,
                                   float64_t* y)
{
    __shared__ float64_t shmem[1024];

    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;
    uint32_t gid = threadIdx.x / 512;
    uint32_t sid = threadIdx.x - 512*gid;

    shmem[tid] = 0.0;

    if(sid >= NUM_SAMPLES) return;

    uint32_t offset = offsetPerBlock[bid] + (segmentsPerBlock[bid]/2)*gid;
    uint32_t nsegments = segmentsPerBlock[bid]/2 + (segmentsPerBlock[bid]%2)*gid;

    uint32_t*  voxel  = voxelIDs + offset;
    uint32_t*  fiber  = fiberIDs + offset;
    uint16_t*  orien  = orienIDs + offset;
    float32_t* length = lengths  + offset;

    float64_t sum = 0.0;

    for(int i = 0; i < nsegments; i++){
        int offset_lut = (*orien)*NUM_SAMPLES + sid;

        float64_t aux = 0.0;
        for(int j = 0; j < NUM_DIAMETERS; j++){
            aux += (double)(lut[offset_lut + j*NUM_ORIENTATIONS*NUM_SAMPLES])*x[(*fiber) + j*NUM_FIBERS];
            //aux += tex1Dfetch(tex_lutIC, offset_lut + j*NUM_ORIENTATIONS*NUM_SAMPLES) * x[(*fiber) + j*NUM_FIBERS];
        }

        sum += aux * (*length);

        fiber++;
        orien++;
        length++;
    }

    shmem[tid] = sum;
    __syncthreads();

    if(tid < NUM_SAMPLES)
        y[(*voxel)*NUM_SAMPLES + sid] = sum + shmem[tid+512];
}

__global__ void multiply_Ax_ECpart(
    uint32_t*  voxelIDs,
    uint16_t*  orienIDs,
    uint32_t*  segmentsPerBlock,
    uint32_t*  offsetPerBlock,
    float32_t* lut,
    float64_t* x,
    float64_t* y)
{
    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;

    if(tid >= NUM_SAMPLES) return;

    uint32_t offset  = offsetPerBlock[bid];
    uint32_t nsegments = segmentsPerBlock[bid];

    uint32_t* voxel = voxelIDs + offset;
    uint16_t* orien = orienIDs + offset;

    uint32_t target = NUM_FIBERS*NUM_DIAMETERS + offset;

    float64_t sum = 0.0;
    for(int i = 0; i < nsegments; i++){
        uint32_t offset_lut = (*orien)*NUM_SAMPLES + tid;

        for(int j = 0; j < NUM_ZEPPELINS; j++)
            sum += (double)(lut[offset_lut + j*NUM_ORIENTATIONS*NUM_SAMPLES])*x[target + j*NUM_PEAKS + i];
            //sum += tex1Dfetch(tex_lutEC, offset_lut + j*NUM_ORIENTATIONS*NUM_SAMPLES) * x[target + j*NUM_PEAKS + i];

        orien++;
    }

    y[(*voxel)*NUM_SAMPLES + tid] += sum;
}

__global__ void multiply_Ax_ISOpart(
    float32_t* lut,
    float64_t* x,
    float64_t* y)
{
    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;

    if(tid >= NUM_SAMPLES) return;

    uint32_t target = NUM_FIBERS*NUM_DIAMETERS + NUM_PEAKS*NUM_ZEPPELINS + bid;

    float64_t sum = 0.0;
    for(int j = 0; j < NUM_BALLS; j++)
        sum += (double)(lut[j*NUM_SAMPLES + tid])*x[target + j*NUM_VOXELS];
        //sum += (double)(tex1Dfetch(tex_lutISO, j*NUM_SAMPLES + tid))*x[target + j*NUM_VOXELS];
        

    y[bid*NUM_SAMPLES + tid] += sum;
}

__global__ void multiply_Aty_ICpart(
    uint32_t*  voxelICt,
    uint32_t*  fiberICt,
    uint16_t*  orienICt,
    float32_t* lengthICt,
    uint32_t*  compartmentsPerBlock,
    uint32_t*  offsetPerBlock,
    float32_t* lut,
    float64_t* x,
    float64_t* y)
{
    __shared__ float64_t shmem[512];

    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;

    shmem[tid] = 0.0;

    if(tid >= NUM_SAMPLES) return;

    uint32_t offset = offsetPerBlock[bid];
    uint32_t nsegments = offset + compartmentsPerBlock[bid];

    uint32_t*  voxel  = voxelICt  + offset;
    uint32_t*  fiber  = fiberICt  + offset;
    uint16_t*  orien  = orienICt  + offset;
    float32_t* length = lengthICt + offset;

    for(int j = 0; j < NUM_DIAMETERS; j++){
        int offset_lut = j*NUM_ORIENTATIONS*NUM_SAMPLES + tid;

        float64_t sum = 0.0;
        voxel  = voxelICt  + offset;
        orien  = orienICt  + offset;
        length = lengthICt + offset;
        for(int i = offset; i < nsegments; i++){
            sum += ((float64_t)(*length)) *( (float64_t) lut[offset_lut + (*orien)*NUM_SAMPLES] )* y[(*voxel)*NUM_SAMPLES + tid];
            //sum += ((float64_t)(*length)) *( (float64_t) tex1Dfetch(tex_lutIC, offset_lut + (*orien)*NUM_SAMPLES) )* y[(*voxel)*NUM_SAMPLES + tid];

            voxel++;
            orien++;
            length++;
        }

        shmem[tid] = sum;
        __syncthreads();

        if(tid < 256) shmem[tid] += shmem[tid + 256]; __syncthreads();
        if(tid < 128) shmem[tid] += shmem[tid + 128]; __syncthreads();
        if(tid <  64) shmem[tid] += shmem[tid +  64]; __syncthreads();
        if(tid <  32) shmem[tid] += shmem[tid +  32]; __syncthreads();
        if(tid <  16) shmem[tid] += shmem[tid +  16]; __syncthreads();
        if(tid <   8) shmem[tid] += shmem[tid +   8]; __syncthreads();
        if(tid <   4) shmem[tid] += shmem[tid +   4]; __syncthreads();

        if(tid == 0) x[j*NUM_FIBERS + (*fiber)] = shmem[0] + shmem[1] + shmem[2] + shmem[3];

        __syncthreads();
    }
}

__global__ void multiply_Aty_ECpart(
    uint32_t*  voxelEC,
    uint16_t*  orienEC,
    uint32_t*  segmentsPerBlock,
    uint32_t*  offsetPerBlock,
    float32_t* lut,
    float64_t* x,
    float64_t* y)
{
    __shared__ float64_t shmem[512];

    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;

    shmem[tid] = 0.0;

    if(tid >= NUM_SAMPLES) return;

    uint32_t offset  = offsetPerBlock[bid];
    uint32_t ncompartments = segmentsPerBlock[bid] + offset;

    uint32_t* voxel = voxelEC + offset;
    uint16_t* orien = orienEC + offset;

    for(int j = 0; j < NUM_ZEPPELINS; j++){        
        uint32_t offset_lut = j*NUM_ORIENTATIONS*NUM_SAMPLES + tid;

        voxel = voxelEC + offset;
        orien = orienEC + offset;
        for(int i = offset; i < ncompartments; i++){
            shmem[tid] =( (float64_t)(lut[(*orien)*NUM_SAMPLES + offset_lut] ))* y[(*voxel)*NUM_SAMPLES + tid];
            //shmem[tid] =( (float64_t)tex1Dfetch(tex_lutEC, (*orien)*NUM_SAMPLES + offset_lut) )* y[(*voxel)*NUM_SAMPLES + tid];
            __syncthreads();

            if(tid < 256) shmem[tid] += shmem[tid + 256]; __syncthreads();
            if(tid < 128) shmem[tid] += shmem[tid + 128]; __syncthreads();
            if(tid <  64) shmem[tid] += shmem[tid +  64]; __syncthreads();
            if(tid <  32) shmem[tid] += shmem[tid +  32]; __syncthreads();
            if(tid <  16) shmem[tid] += shmem[tid +  16]; __syncthreads();
            if(tid <   8) shmem[tid] += shmem[tid +   8]; __syncthreads();
            if(tid <   4) shmem[tid] += shmem[tid +   4]; __syncthreads();
            if(tid <   2) shmem[tid] += shmem[tid +   2]; __syncthreads();

            if(tid == 0) x[NUM_FIBERS*NUM_DIAMETERS + j*NUM_PEAKS + i] = shmem[0] + shmem[1];

            voxel++;
            orien++;
            __syncthreads();
        }
    }
}

__global__ void multiply_Aty_ISOpart(float* lut, double* x, double* y){
    __shared__ double shmem[512];

    uint bid = blockIdx.x;
    uint tid = threadIdx.x;
    uint offset = NUM_FIBERS*NUM_DIAMETERS + NUM_PEAKS*NUM_ZEPPELINS + bid;

    shmem[tid] = 0.0;

    if(tid >= NUM_SAMPLES) return;

    for(int j = 0; j < NUM_BALLS; j++){
        shmem[tid] =( (float64_t) lut[j*NUM_SAMPLES + tid] )* y[bid*NUM_SAMPLES + tid];
        //shmem[tid] =( (float64_t) tex1Dfetch(tex_lutISO, j*NUM_SAMPLES + tid) )* y[bid*NUM_SAMPLES + tid];
        __syncthreads();

        if(tid < 256) shmem[tid] += shmem[tid + 256]; __syncthreads();
        if(tid < 128) shmem[tid] += shmem[tid + 128]; __syncthreads();
        if(tid <  64) shmem[tid] += shmem[tid +  64]; __syncthreads();
        if(tid <  32) shmem[tid] += shmem[tid +  32]; __syncthreads();
        if(tid <  16) shmem[tid] += shmem[tid +  16]; __syncthreads();
        if(tid <   8) shmem[tid] += shmem[tid +   8]; __syncthreads();
        if(tid <   4) shmem[tid] += shmem[tid +   4]; __syncthreads(); 

        if(tid == 0)
            x[offset + j*NUM_VOXELS] = shmem[0] + shmem[1] + shmem[2] + shmem[3];
    }
}

